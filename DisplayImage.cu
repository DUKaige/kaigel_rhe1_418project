#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <string>
#include <sstream>

using namespace std;

#define kernel_width 7
#define threadsPerBlock 512
#define NUM_ITER_DFS 3

float stoff(const char* s){
  float rez = 0, fact = 1;
  if (*s == '-'){
    s++;
    fact = -1;
  };
  for (int point_seen = 0; *s; s++){
    if (*s == '.'){
      point_seen = 1; 
      continue;
    };
    int d = *s - '0';
    if (d >= 0 && d <= 9){
      if (point_seen) fact /= 10.0f;
      rez = rez * 10.0f + (float)d;
    };
  };
  return rez * fact;
}

__global__ void kernel_blur(float* pixels, float* output, int width, int height, int N) {
    const float kernel[kernel_width][kernel_width] = {
        {0.00000067, 0.00002292, 0.00019117, 0.00038771, 0.00019117, 0.00002292, 0.00000067},
        {0.00002292, 0.00078633, 0.00655965, 0.01330373, 0.00655965, 0.00078633, 0.00002292},
        {0.00019117, 0.00655965, 0.05472157, 0.11098164, 0.05472157, 0.00655965, 0.00019117},
        {0.00038771, 0.01330373, 0.11098164, 0.22508352, 0.11098164, 0.01330373, 0.00038771},
        {0.00019117, 0.00655965, 0.05472157, 0.11098164, 0.05472157, 0.00655965, 0.00019117},
        {0.00002292, 0.00078633, 0.00655965, 0.01330373, 0.00655965, 0.00078633, 0.00002292},
        {0.00000067, 0.00002292, 0.00019117, 0.00038771, 0.00019117, 0.00002292, 0.00000067}
    };

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N) {
        int row = index / width;
        int col = index % width;
        float sum = 0;
        float denom = 0;
        int rowStart = row - kernel_width/2;
        int rowEnd = row + kernel_width/2 + 1;
        int colStart = col - kernel_width/2;
        int colEnd = col + kernel_width/2 + 1;
        for (int smallRow = rowStart; smallRow < rowEnd; smallRow ++) {
            for (int smallCol = colStart; smallCol < colEnd; smallCol ++) {
                if (smallRow >= 0 && smallRow < height && smallCol >= 0 && smallCol < width) {
                    sum += kernel[smallRow - rowStart][smallCol - colStart] * pixels[smallRow * width + smallCol];
                    denom += kernel[smallRow - rowStart][smallCol - colStart];
                }
            }
        }
        output[index] = sum/denom;
    }   
}

__global__ void kernel_calculateGradient(float* pixelsAfterBlur, float* gradientMag, int* gradientAng, int width, int height, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N) {
        int row = index / width;
        int col = index % width;
        float gy = (float) row == height - 1? 0 : pixelsAfterBlur[index + width] - pixelsAfterBlur[index];
        float gx = (float) col == width - 1? 0 : pixelsAfterBlur[index + 1] - pixelsAfterBlur[index];
        gradientMag[index] = sqrt(gx * gx + gy * gy);
        float ang;
        if (gx < 0.000001 && gx > -0.000001) ang = 90;
        else ang = atan(gy / gx) / 3.1415926 * 180.0;
        if (ang < 0)
            ang += 180;
        gradientAng[index] = ((int) (ang + 22.5) / 45) * 45;
    }
}

__global__ void kernel_doubleThreshold(float* pixelsAfterThin, int* pixelsStrongEdges, int* pixelsWeakEdges, int width, int height, float low_threshold, float high_threshold, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N) {
        float val = pixelsAfterThin[index];
        if (val >= high_threshold){
            pixelsStrongEdges[index] = 1;
        }
        if (val < high_threshold && val >= low_threshold){
            pixelsWeakEdges[index] = 1;
        }
    }
}

__global__ void kernel_thin(float* pixelsAfterThin, int* gradientAng, float* gradientMag, int width, int height, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N) {
        int row = index / width;
        int col = index % width;
        float mag = gradientMag[row * width + col];
        float magL = 0;
        float magR = 0;
        int ang = gradientAng[row * width + col];
        if (ang == 0 || ang == 180) {
            if (row > 0) magL = gradientMag[row * width + col - 1];
            if (row < height - 1) magR = gradientMag[row * width + col + 1];
        } 

        else if (ang == 45 || ang == 225) {
            if (row > 0 && col < width - 1) magL = gradientMag[(row + 1) * width + col + 1];

            if (row < height - 1 && col > 0) magR = gradientMag[(row - 1) * width + col - 1];
        } 

        else if (ang == 90 || ang == 270) {
            if (col > 0) magL = gradientMag[(row - 1) * width + col];

            if (col < width - 1) magR = gradientMag[(row + 1) * width + col];
        } 

        else if (ang == 135 || ang == 315) {
            if (row > 0 && col > 0) magL = gradientMag[(row + 1) * width + col - 1];

            if (row < height - 1 && col < width - 1) magR = gradientMag[(row - 1) * width + col + 1];
        }
        if (mag > magL && mag > magR) {
            pixelsAfterThin[row * width + col] = mag;
        } 
        else {
            pixelsAfterThin[row * width + col] = 0;
        }
    }
}


void blur(float* pixels, float* output, int width, int height, int N, int blocks) {
    float* cudaPixels;
    float* cudaOutput;
    hipMalloc(&cudaPixels, N * sizeof(float));
    hipMalloc(&cudaOutput, N * sizeof(float));
    hipMemcpy(cudaPixels, pixels, N * sizeof(float), hipMemcpyHostToDevice);
    kernel_blur<<<blocks, threadsPerBlock>>>(cudaPixels, cudaOutput, width, height, N);
    hipDeviceSynchronize();
    hipMemcpy(output, cudaOutput, N * sizeof(float), hipMemcpyDeviceToHost);
    //hipFree(cudaPixels);
    //hipFree(cudaOutput);
}

void calculateGradient(float* pixelsAfterBlur, float* gradientMag, int* gradientAng, int width, int height, float* maxMag, int N, int blocks) {
    float* cudaPixels;
    float* cudaGradientMag;
    int* cudaGradientAng;
    hipMalloc(&cudaPixels, N * sizeof(float));
    hipMalloc(&cudaGradientAng, N * sizeof(int));
    hipMalloc(&cudaGradientMag, N * sizeof(float));
    hipMemcpy(cudaPixels, pixelsAfterBlur, N * sizeof(float), hipMemcpyHostToDevice);
    kernel_calculateGradient<<<blocks, threadsPerBlock>>>(cudaPixels, cudaGradientMag, cudaGradientAng, width, height, N);
    hipDeviceSynchronize();
    hipMemcpy(gradientMag, cudaGradientMag, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(gradientAng, cudaGradientAng, N * sizeof(int), hipMemcpyDeviceToHost);


    float max = 0;
    for (int i = 0; i < N; i++) {
        if (gradientMag[i] > max) {
            max = gradientMag[i];
        }
    }
    *maxMag = max;
    hipFree(cudaPixels);
    hipFree(cudaGradientAng);
    hipFree(cudaGradientMag);
}

void thin(float* gradientMag, int* gradientAng, float* pixelsAfterThin, int width, int height, int N, int blocks) {
    float* cudaPixels;
    float* cudaGradientMag;
    int* cudaGradientAng;
    hipMalloc(&cudaPixels, N * sizeof(float));
    hipMalloc(&cudaGradientAng, N * sizeof(int));
    hipMalloc(&cudaGradientMag, N * sizeof(float));
    hipMemcpy(cudaPixels, pixelsAfterThin, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(cudaGradientAng, gradientAng, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(cudaGradientMag, gradientMag, N * sizeof(float), hipMemcpyHostToDevice);
    kernel_thin<<<blocks, threadsPerBlock>>>(cudaPixels, cudaGradientAng, cudaGradientMag, width, height, N);
    hipDeviceSynchronize();
    hipMemcpy(pixelsAfterThin, cudaPixels, N * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(cudaPixels);
    hipFree(cudaGradientMag);
    hipFree(cudaGradientAng);
}

void doubleThreshold(float* pixelsAfterThin, int* pixelsStrongEdges, int* pixelsWeakEdges, int width, int height, float low_threshold, float high_threshold, int N, int blocks) {
    float* cudaPixels;
    int* cudaStrongEdges;
    int* cudaWeakEdges;
    hipMalloc(&cudaPixels, N * sizeof(float));
    hipMalloc(&cudaStrongEdges, N * sizeof(int));
    hipMalloc(&cudaWeakEdges, N * sizeof(int));
    hipMemcpy(cudaPixels, pixelsAfterThin, N * sizeof(float), hipMemcpyHostToDevice);
    kernel_doubleThreshold<<<blocks, threadsPerBlock>>>(cudaPixels, cudaStrongEdges, cudaWeakEdges, width, height, low_threshold, high_threshold, N);
    hipDeviceSynchronize();
    hipMemcpy(pixelsStrongEdges, cudaStrongEdges, N * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(pixelsWeakEdges, cudaWeakEdges, N * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(cudaPixels);
    hipFree(cudaStrongEdges);
    hipFree(cudaWeakEdges);
}

__device__ __inline__ void dfsRange(int row, int col, int lorow, int hirow, int locol, int hicol, int* pixelsStrongEdges, int* pixelsWeakEdges, int* visited, int width, int height) {
    vector<int> stack;
    int idx = row * width + col;
    stack.push_back(idx);
    while (!stack.empty()) {
        idx = stack.back();
        stack.pop_back();
        if (pixelsWeakEdges[idx]) {
            pixelsStrongEdges[idx] = 1;
        }
        int id;
        if (pixelsStrongEdges[idx]) {
            if (row > lorow) {
                id = (row - 1) * width + col;
                if (!visited[id]) {
                    stack.push_back(id);
                    visited[id] = 1;
                }
                if (col > locol) {
                    id = (row - 1) * width + col - 1;
                    if (!visited[id]){
                        stack.push_back(id);
                        visited[id] = 1;
                    } 
                }

                if (col < hicol - 1) {
                    id = (row - 1) * width + col + 1;
                    if (!visited[id]) {
                        stack.push_back(id);
                        visited[id] = 1;
                    } 
                }
            }

            if (row < hirow - 1) {
                id = (row + 1) * width + col;
                if (!visited[id]) {
                    stack.push_back(id);
                    visited[id] = 1;
                }
                if (col > locol) {
                    id = (row + 1) * width + col - 1;
                    if (!visited[id]){
                        stack.push_back(id);
                        visited[id] = 1;
                    } 
                }

                if (col < hicol - 1) {
                    id = (row + 1) * width + col + 1;
                    if (!visited[id]) {
                        stack.push_back(id);
                        visited[id] = 1;
                    }
                }
            }

            if (col > locol) {
                id = row * width + col - 1;
                if (!visited[id]) {
                    stack.push_back(id);
                    visited[id] = 1;
                }            
            }

            if (col < hicol - 1) {
                id = row * width + col + 1;
                if (!visited[id]) {
                    stack.push_back(id);
                    visited[id] = 1;
                }               
            }
        }
    }    
}

__global__ void kernel_dfs(int numDiv, int* pixelsStrongEdges, int* pixelsWeakEdges, int* visited, int width, int height) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= numDiv * numDiv) return;
    int colIndex = index % numDiv;
    int rowIndex = index / numDiv;
    int colStart = colIndex * width / numDiv;
    int colEnd = (colIndex + 1) * width / numDiv;
    int rowStart = rowIndex * width / numDiv;
    int rowEnd = (rowIndex + 1) * width / numDiv;


    for (int row = rowStart; row < rowEnd; row ++) {
        for (int col = colStart; col < colEnd; col ++) {
            if (pixelsStrongEdges[row * width + col] == 1)
                dfsRange(row, col, 0, height, 0, width, pixelsStrongEdges, pixelsWeakEdges, visited, width, height);

        }
    }
}


__global__ void kernel_exchange(int numDiv, int* pixelsStrongEdges, int* pixelsWeakEdges, int* visited, int width, int height) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= numDiv * numDiv) return;
    int colIndex = index % numDiv;
    int rowIndex = index / numDiv;
    int colStart = colIndex * width / numDiv;
    int colEnd = (colIndex + 1) * width / numDiv;
    int rowStart = rowIndex * width / numDiv;
    int rowEnd = (rowIndex + 1) * width / numDiv;
    // Left
    if (colStart > 0) {
        for (int row = rowStart; row < rowEnd; row ++) {
            if (pixelsStrongEdges[row * width + colStart] == 1) {
                pixelsStrongEdges[row * width + colStart - 1] = 1;
            }
        }
    }

    // Right
    if (colEnd < width) {
        for (int row = rowStart; row < rowEnd; row ++) {
            if (pixelsStrongEdges[row * width + colEnd - 1] == 1) {
                pixelsStrongEdges[row * width + colEnd] = 1;
            }
        }
    }

    // Top
    if (rowStart > 0) {
        for (int col = colStart; col < colEnd; col ++) {
            if (pixelsStrongEdges[rowStart * width + col] == 1) {
                pixelsStrongEdges[(rowStart - 1) * width + col] = 1;
            }
        }
    }

    // Bottom
    if (rowEnd < height) {
        for (int col = colStart; col < colEnd; col ++) {
            if (pixelsStrongEdges[(rowEnd - 1) * width + col] == 1) {
                pixelsStrongEdges[rowEnd * width + col] = 1;
            }
        }
    }
}

void edgeTrack(int* pixelsStrongEdges, int* pixelsWeakEdges, int width, int height) {
    int* visited = (int*) calloc(sizeof(int), width * height);
    int numDiv = min(min(256, height/16), width/16);
    int blocks = (numDiv * numDiv + threadsPerBlock - 1) / threadsPerBlock;
    for (int i = 0; i < NUM_ITER_DFS; i ++) {
        kernel_exchange(numDiv, pixelsStrongEdges, pixelsWeakEdges, visited, width, height);
        kernel_dfs<<<blocks, threadsPerBlock>>>(numDiv, pixelsStrongEdges, pixelsWeakEdges, visited, width, height);
    }

}

float* split(string str, char delimiter, int numElts) {
    float* elts = (float*) malloc(sizeof(float) * numElts);
    stringstream ss(str);
    string tok;
    int i = 0; 

    while(getline(ss, tok, delimiter)) {
        elts[i++] = stoff(tok.c_str());
    }
 
    return elts;
}

int main(int argc, char** argv) {  

    if (argc != 2) {
        printf("usage: DisplayImage.out <Image_Path>\n");
            return -1;
    }

    float low_threshold = 0.05;
    float high_threshold = 0.1;
    float* pixels;
    int height;
    int width;

    string line;
    ifstream myfile (argv[1]);
    if (myfile.is_open()) {

        getline(myfile, line);
        height = stoff(line.c_str());
        getline(myfile, line);
        width = stoff(line.c_str());

        pixels = (float*) malloc(sizeof(float) * height * width);
        int idx = 0;
        while (getline(myfile, line)) {
            float* content = split(line, ' ', width);
            memcpy(pixels+idx, content, sizeof(float) * width);
            idx += width;
            free(content);
        }
        myfile.close();
    } 
    else {
        printf("Unable to open file"); 
        return -1;
    }

    int N = height * width;
    int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;

    // /* 1. blur */
    float* pixelsAfterBlur = (float*) malloc(sizeof(float)*height*width);
    blur(pixels, pixelsAfterBlur, width, height, N, blocks);

    /* 2. gradient */
    float* gradientMag = (float*) malloc(sizeof(float)*height*width);
    int* gradientAng = (int*) malloc(sizeof(int)*height*width);
    float maxMag = -1;
    calculateGradient(pixelsAfterBlur, gradientMag, gradientAng, width, height, &maxMag, N, blocks);

    /* 3. non-maximum suppresion */
    float* pixelsAfterThin = (float*) malloc(sizeof(float)*height*width);
    thin(gradientMag, gradientAng, pixelsAfterThin, width, height, N, blocks);

    /* 4. double thresholding */
    int* pixelsStrongEdges = (int*) calloc(sizeof(int), height*width);
    int* pixelsWeakEdges = (int*) calloc(sizeof(int), height*width);
    doubleThreshold(pixelsAfterThin, pixelsStrongEdges, pixelsWeakEdges, width, height, low_threshold * maxMag, high_threshold * maxMag, N, blocks);

    /* 5. edge tracking */
    edgeTrack(pixelsStrongEdges, pixelsWeakEdges, width, height);


    /* 6. display */
    ofstream outfile ("result.txt");
    if (outfile.is_open()) {

        outfile << height << "\n";
        outfile << width << "\n";
        int idx = 0;
        for (int i = 0; i < height; i++) {
            for (int j = 0; j < width; j++) {
                outfile <<  pixelsStrongEdges[idx++] * 255 << " ";
            }
            outfile << "\n";
        }
        outfile.close();
    }

    return 0;
}